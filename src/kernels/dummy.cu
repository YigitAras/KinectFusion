#include "hip/hip_runtime.h"
#include "dummy.cuh"
#include <iostream>
#include <opencv2/core/cuda.hpp>
#include "cuda_error_handle.h"
#include "Volume.h"
#include <assert.h>
#include <stdio.h>

#define assert(X)                                                    \
	if (!(X))                                                        \
		printf("tid %d: %s, %d\n", threadIdx.x, __FILE__, __LINE__); \
	return;

#define ICP_DISTANCE_THRESHOLD 0.01f // inspired from excellence in m
// The angle threshold (as described in the paper) in degrees
#define ICP_ANGLE_THRESHOLD 20.f // inspired from excellence in degrees
#define VOXSIZE 0.01f			 // in m
// TODO: hardcoded in multiple places
#define MIN_DEPTH 0.2f		   //in m
#define DISTANCE_THRESHOLD 2.f // inspired
#define MAX_WEIGHT_VALUE 128.f //inspired
#define ICP_ITERATIONS 10
__global__ void updateReconstructionKernel(
	Eigen::Matrix<int, 3, 1, Eigen::DontAlign> gridSize,
	cv::cuda::PtrStepSzf volume,
	CameraParameters cameraParams,
	cv::cuda::PtrStepSzf depthMap,
	Eigen::Matrix<float, 4, 4, Eigen::DontAlign> poseInverse,
	float minf)
{

	unsigned int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	unsigned int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	unsigned int lim = gridSize.x() *
					   gridSize.y() *
					   gridSize.z();

	//assert(gridSize.x() == 512);
	const Eigen::Matrix<float, 3, 1, Eigen::DontAlign> translation = poseInverse.block<3, 1>(0, 3);

	const Eigen::Matrix<float, 3, 3, Eigen::DontAlign> rotation = poseInverse.block<3, 3>(0, 0);

	if (x >= 0 && x < gridSize.x() &&
		y >= 0 && y < gridSize.y())
	{
		for (auto z = 0; z < gridSize.z(); z++)
		{
			// TODO: Why now unsigned long long to avoid overflow?
			int ind = (x * gridSize.y() + y) * gridSize.z() + z;
			if (ind < lim)
			{
				// printf("%d %d %d \n",x,y,z);
				int vx = x - ((gridSize.x() - 1) / 2);
				int vy = y - ((gridSize.y() - 1) / 2);
				int vz = z - ((gridSize.z() - 1) / 2);
				Vector3f voxelWorldPosition(vx + 0.5, vy + 0.5, vz + 0.5);
				voxelWorldPosition *= VOXSIZE; //TODO: //model.voxSize;

				Eigen::Matrix<float, 3, 1, Eigen::DontAlign> voxelCamPosition = rotation * voxelWorldPosition + translation;
				// voxelCamPosition = voxelCamPosition + translation;

				if (voxelCamPosition.z() < 0)
				{
					continue;
				}

				const Vector2i imagePosition(
					(voxelCamPosition.y() / voxelCamPosition.z()) * cameraParams.fovY + cameraParams.cY,
					(voxelCamPosition.x() / voxelCamPosition.z()) * cameraParams.fovX + cameraParams.cX);

				if (!(imagePosition.x() < 0 ||
					  imagePosition.x() >= cameraParams.depthImageHeight ||
					  imagePosition.y() < 0 ||
					  imagePosition.y() >= cameraParams.depthImageWidth))
				{
					// const float depth = depthMap[imagePosition.x() * cameraParams.depthImageWidth + imagePosition.y()];
					const float depth = depthMap(imagePosition.x(), imagePosition.y());
					if (depth > 0 && depth != minf)
					{

						const Vector3f homogenImagePosition(
							(imagePosition.x() - cameraParams.cX) / cameraParams.fovX,
							(imagePosition.y() - cameraParams.cY) / cameraParams.fovY,
							1.0f);
						const float lambda = homogenImagePosition.norm();

						const float value = (-1.f) * ((1.f / lambda) * (voxelCamPosition).norm() - depth);
						if (value >= -DISTANCE_THRESHOLD)
						{

							const float sdfValue = fmin(1.f, value / DISTANCE_THRESHOLD);

							const float currValue = volume(ind, 0);
							const float currWeight = volume(ind, 1);

							const float addWeight = 1;
							const float nextTSDF =
								(currWeight * currValue + addWeight * sdfValue) /
								(currWeight + addWeight);
							// TODO: Check the MAX_WEIGHT_VALUE and how it would work after max iterations
							volume(ind, 0) = nextTSDF;
							volume(ind, 1) = fmin(currWeight + addWeight, MAX_WEIGHT_VALUE);
						}
					}
				}
			}
		}
	}
}

__device__ bool isValid(Eigen::Matrix<int, 3, 1, Eigen::DontAlign> gridSize,
						Eigen::Matrix<float, 3, 1, Eigen::DontAlign> point)
{
	return point.x() < gridSize.x() / 2 && point.y() < gridSize.y() / 2 &&
		   point.z() < gridSize.z() / 2 && point.x() > -gridSize.x() / 2 &&
		   point.y() > -gridSize.y() / 2 && point.z() > -gridSize.z() / 2;
}

// @position should be in voxelCoordinates [-something, something]
__device__ float getFromVolume(cv::cuda::PtrStepSzf volume,
							   Eigen::Matrix<float, 3, 1, Eigen::DontAlign> position,
							   Eigen::Matrix<int, 3, 1, Eigen::DontAlign> gridSize)
{
	unsigned int vx = position.x() + ((gridSize.x() - 1) / 2);
	unsigned int vy = position.y() + ((gridSize.y() - 1) / 2);
	unsigned int vz = position.z() + ((gridSize.z() - 1) / 2);

	unsigned int ind = (vx * gridSize.y() + vy) * gridSize.z() + vz;
	// printf("%f -> ;) (%d, %d, %d)\n", volume(ind, 0), vx, vy, vz);
	return volume(ind, 0);
}

__device__ float interpolation(cv::cuda::PtrStepSzf volume,
							   Eigen::Matrix<float, 3, 1, Eigen::DontAlign> position,
							   Eigen::Matrix<int, 3, 1, Eigen::DontAlign> gridSize)
{
	Vector3f pointInGrid((int)position.x(), (int)position.y(), (int)position.z());

	// Toggle to disable interpolation
	//}	return getFromVolume(volume, pointInGrid, gridSize);

	Vector3f voxelCenter(pointInGrid.x() + 0.5f, pointInGrid.y() + 0.5f,
						 pointInGrid.z() + 0.5f);

	pointInGrid.x() = (position.x() < voxelCenter.x()) ? (pointInGrid.x() - 1)
													   : pointInGrid.x();
	pointInGrid.y() = (position.y() < voxelCenter.y()) ? (pointInGrid.y() - 1)
													   : pointInGrid.y();
	pointInGrid.z() = (position.z() < voxelCenter.z()) ? (pointInGrid.z() - 1)
													   : pointInGrid.z();

	// pointInGrid = Vector3f(pointInGrid.x() - 1, pointInGrid.y() - 1,
	// pointInGrid.z() - 1);

	// Check Distance correctness
	const float distX = abs((abs(position.x()) - abs((pointInGrid.x()) + 0.5f)));
  	const float distY = abs((abs(position.y()) - abs((pointInGrid.y()) + 0.5f)));
  	const float distZ = abs((abs(position.z()) - abs((pointInGrid.z()) + 0.5f)));

	// TODO: Check the correctness of below, just a sanity check
	return (isValid(gridSize, pointInGrid)
				? getFromVolume(volume, pointInGrid, gridSize)
				: 0.0f) *
			   (1 - distX) * (1 - distY) * (1 - distZ) +
		   (isValid(gridSize,
					Vector3f(pointInGrid.x(), pointInGrid.y(), pointInGrid.z() + 1))
				? getFromVolume(volume, Vector3f(pointInGrid.x(), pointInGrid.y(), pointInGrid.z() + 1), gridSize)
				: 0.0f) *
			   (1 - distX) * (1 - distY) * (distZ) +
		   (isValid(gridSize,
					Vector3f(pointInGrid.x(), pointInGrid.y() + 1, pointInGrid.z()))
				? getFromVolume(volume, Vector3f(pointInGrid.x(), pointInGrid.y() + 1, pointInGrid.z()), gridSize)
				: 0.0f) *
			   (1 - distX) * distY * (1 - distZ) +
		   (isValid(gridSize, Vector3f(pointInGrid.x(), pointInGrid.y() + 1,
									   pointInGrid.z() + 1))
				? getFromVolume(volume, Vector3f(pointInGrid.x(), pointInGrid.y() + 1, pointInGrid.z() + 1), gridSize)
				: 0.0f) *
			   (1 - distX) * distY * distZ +
		   (isValid(gridSize,
					Vector3f(pointInGrid.x() + 1, pointInGrid.y(), pointInGrid.z()))
				? getFromVolume(volume, Vector3f(pointInGrid.x() + 1, pointInGrid.y(), pointInGrid.z()), gridSize)
				: 0.0f) *
			   distX * (1 - distY) * (1 - distZ) +
		   (isValid(gridSize, Vector3f(pointInGrid.x() + 1, pointInGrid.y(),
									   pointInGrid.z() + 1))
				? getFromVolume(volume, Vector3f(pointInGrid.x() + 1, pointInGrid.y(), pointInGrid.z() + 1), gridSize)
				: 0.0f) *
			   distX * (1 - distY) * distZ +
		   (isValid(gridSize, Vector3f(pointInGrid.x() + 1, pointInGrid.y() + 1,
									   pointInGrid.z()))
				? getFromVolume(volume, Vector3f(pointInGrid.x() + 1, pointInGrid.y() + 1, pointInGrid.z()), gridSize)
				: 0.0f) *
			   distX * distY * (1 - distZ) +
		   (isValid(gridSize, Vector3f(pointInGrid.x() + 1, pointInGrid.y() + 1,
									   pointInGrid.z() + 1))
				? getFromVolume(volume, Vector3f(pointInGrid.x() + 1, pointInGrid.y() + 1, pointInGrid.z() + 1), gridSize)
				: 0.0f) *
			   distX * distY * distZ;
}

// TODO: interpolation

__global__ void rayCastKernel(Eigen::Matrix<float, 4, 4, Eigen::DontAlign> cameraPose,
							  CameraParameters params,
							  Eigen::Matrix<int, 3, 1, Eigen::DontAlign> gridSize,
							  cv::cuda::PtrStepSzf volume,
							  cv::cuda::PtrStepSz<float3> surfacePoints,
							  cv::cuda::PtrStepSz<float3> surfaceNormals)
{

	unsigned int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	unsigned int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (0 <= x && x <= params.depthImageWidth && 0 <= y && y <= params.depthImageHeight)
	{

		// surfacePoints(x, y) = make_float3(255.0, 255.0, 255.0);
		const Vector3f pixelInCameraCoords((x - params.cX) / params.fovX,
										   (y - params.cY) / params.fovY, 1.0);

		Vector3f currPositionInCameraWorld = pixelInCameraCoords.normalized() * MIN_DEPTH;

		currPositionInCameraWorld += cameraPose.block<3, 1>(0, 3);
		Vector3f rayStepVec = pixelInCameraCoords.normalized() * VOXSIZE;
		// Rotate rayStepVec to 3D world
		rayStepVec = (cameraPose.block<3, 3>(0, 0) * rayStepVec);

		Vector3f voxelInGridCoords = currPositionInCameraWorld / VOXSIZE;
		Vector3f currPoint, currNormal;

		float currTSDF = 1.0;
		bool sign = true;
		bool prevSign = sign;

		int maxRayDist = 1000;

		while ((prevSign == sign) && isValid(gridSize, voxelInGridCoords))
		{
			currTSDF = getFromVolume(volume, voxelInGridCoords, gridSize);

			voxelInGridCoords = currPositionInCameraWorld / VOXSIZE;
			currPositionInCameraWorld += rayStepVec;

			prevSign = sign;
			sign = currTSDF >= 0;
		}
		// printf("OUT");
		if ((sign != prevSign) && isValid(gridSize, voxelInGridCoords))
		{
			currPoint = currPositionInCameraWorld;
		}
		else
		{
			return;
		}

		Vector3f neighbor = voxelInGridCoords;
		neighbor.x() += 1;
		if (!isValid(gridSize, neighbor))
			return;
		const float Fx1 = interpolation(volume, neighbor, gridSize);

		neighbor = voxelInGridCoords;

		neighbor.x() -= 1;
		if (!isValid(gridSize, neighbor))
			return;
		const float Fx2 = interpolation(volume, neighbor, gridSize);

		currNormal.x() = (Fx1 - Fx2);

		neighbor = voxelInGridCoords;
		neighbor.y() += 1;
		if (!isValid(gridSize, neighbor))
			return;
		const float Fy1 = interpolation(volume, neighbor, gridSize);

		neighbor = voxelInGridCoords;
		neighbor.y() -= 1;
		if (!isValid(gridSize, neighbor))
			return;
		const float Fy2 = interpolation(volume, neighbor, gridSize);

		currNormal.y() = (Fy1 - Fy2);

		neighbor = voxelInGridCoords;
		neighbor.z() += 1;
		if (!isValid(gridSize, neighbor))
			return;
		const float Fz1 = interpolation(volume, neighbor, gridSize);

		neighbor = voxelInGridCoords;
		neighbor.z() -= 1;
		if (!isValid(gridSize, neighbor))
			return;
		const float Fz2 = interpolation(volume, neighbor, gridSize);

		currNormal.z() = (Fz1 - Fz2);

		if (currNormal.norm() == 0)
			return;

		currNormal.normalize();
		surfacePoints(y, x) = make_float3(currPoint.x(), currPoint.y(), currPoint.z());
		// printf("%f %f %f \n ",currPoint.x(), currPoint.y(), currPoint.z());
		surfaceNormals(y, x) = make_float3(currNormal.x(), currNormal.y(), currNormal.z());

		// return true;
	}
	// TODO: set the value for surface point and normal
	// bool exists =
	// 		pointRay(cameraPose, params, y, x, currPoint, currNormal);
	// if (exists)
	// {
	// 	surfacePoints.push_back(currPoint);
	// 	surfaceNormals.push_back(currNormal);
	// }
}

__global__ void findCorrespondencesKernel(Eigen::Matrix<float, 4, 4, Eigen::DontAlign> modelToFrameInverse,
										  Eigen::Matrix<float, 4, 4, Eigen::DontAlign> estimatedCameraPose,
										  CameraParameters cameraParams,
										  cv::cuda::PtrStepSz<float3> surfacePoints,
										  cv::cuda::PtrStepSz<float3> surfaceNormals,
										  cv::cuda::PtrStepSz<float3> newVertexMap,
										  cv::cuda::PtrStepSz<float3> newNormalMap,
										  cv::cuda::PtrStepSz<int2> matches)
{

	unsigned int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	unsigned int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	Eigen::Matrix<float, 3, 1, Eigen::DontAlign> n, d, s;

	Eigen::Matrix<float, 4, 4, Eigen::DontAlign> estimatedFrameToFrame = modelToFrameInverse * estimatedCameraPose;
	// printf("%f %f %f \n",estimatedCameraPose(0,0),estimatedCameraPose(1,1),estimatedCameraPose(2,2));
	const Eigen::Matrix<float, 3, 1, Eigen::DontAlign> estimatedFrametoFrameTranslation = estimatedFrameToFrame.block<3, 1>(0, 3);
	const Eigen::Matrix<float, 3, 3, Eigen::DontAlign> estimatedFrameToFrameRotation = estimatedFrameToFrame.block<3, 3>(0, 0);

	const Eigen::Matrix<float, 3, 1, Eigen::DontAlign> estimatedModelToFrameTranslation = estimatedCameraPose.block<3, 1>(0, 3);
	const Eigen::Matrix<float, 3, 3, Eigen::DontAlign> estimatedModelToFrameRotation = estimatedCameraPose.block<3, 3>(0, 0);

	if (0 <= x && x <= cameraParams.depthImageWidth && 0 <= y && y <= cameraParams.depthImageHeight)
	{
		Eigen::Matrix<float, 3, 1, Eigen::DontAlign> newNormal;
		newNormal.x() = newNormalMap(y, x).x;
		newNormal.y() = newNormalMap(y, x).y;
		newNormal.z() = newNormalMap(y, x).z;

		if (!(newNormal.x() == 0 &&
			  newNormal.y() == 0 &&
			  newNormal.z() == 0))
		{
			Eigen::Matrix<float, 3, 1, Eigen::DontAlign> newVertex;
			newVertex.x() = newVertexMap(y, x).x;
			newVertex.y() = newVertexMap(y, x).y;
			newVertex.z() = newVertexMap(y, x).z;
			//pose is passed as inverse
			Eigen::Matrix<float, 3, 1, Eigen::DontAlign> newVertexCamera = estimatedFrameToFrameRotation * newVertex + estimatedFrametoFrameTranslation;
			Eigen::Matrix<float, 3, 1, Eigen::DontAlign> newVertexGlobal = estimatedModelToFrameRotation * newVertex + estimatedModelToFrameTranslation;

			//we do this part differently since here there is no iterative update step

			Eigen::Vector2i point;
			//TODO this is stolen
			point.x() = (int)(newVertexCamera.x() * cameraParams.fovX / newVertexCamera.z() + cameraParams.cX + 0.5f);
			point.y() = (int)(newVertexCamera.y() * cameraParams.fovY / newVertexCamera.z() + cameraParams.cY + 0.5f);
			if (point.x() >= 0 && point.y() >= 0 &&
				point.x() < cameraParams.depthImageWidth &&
				point.y() < cameraParams.depthImageHeight &&
				newVertexCamera.z() >= 0)
			{
				Eigen::Matrix<float, 3, 1, Eigen::DontAlign> oldNormal;
				oldNormal.x() = surfaceNormals(point.y(), point.x()).x;
				oldNormal.y() = surfaceNormals(point.y(), point.x()).y;
				oldNormal.z() = surfaceNormals(point.y(), point.x()).z;
				if (!(oldNormal.x() == 0 &&
					  oldNormal.y() == 0 &&
					  oldNormal.z() == 0))
				{
					Eigen::Matrix<float, 3, 1, Eigen::DontAlign> oldVertex;

					oldVertex.x() = surfacePoints(point.y(), point.x()).x;
					oldVertex.y() = surfacePoints(point.y(), point.x()).y;
					oldVertex.z() = surfacePoints(point.y(), point.x()).z;
					const float distance = (oldVertex - newVertexGlobal).norm();
					if (distance <= ICP_DISTANCE_THRESHOLD)
					{

						Eigen::Matrix<float, 3, 1, Eigen::DontAlign> newNormalGlobal = estimatedModelToFrameRotation * newNormal;
						const float sine = newNormalGlobal.cross(oldNormal).norm() *180.f/M_PI;

						if (sine >= ICP_ANGLE_THRESHOLD)
						{
							// n = oldNormal;
							// d = oldVertex;
							// s = newVertex;
							//TODO : Make sure this is correct accessing

							// printf("%d %d  matched with %d %d     sine %f \n",y,x,point.y(),point.x(),sine);
							matches(y, x) = make_int2(point.y(), point.x());
						}
					}
				}
			}
		}
	}
}
namespace Wrapper
{
	void updateReconstruction(Volume &model,
							  const CameraParameters &cameraParams,
							  const float *const depthMap,
							  const MatrixXf &poseInverse)
	{
		std::vector<int> sizes{model.gridSize.x(), model.gridSize.y(),
							   model.gridSize.z()};
		cv::cuda::GpuMat deviceModel; //(sizes,CV_32FC2);

		// TODO: Find better optimization for GPU Arch
		const int threadsX = 1, threadsY = 1;
		const dim3 threads(threadsX, threadsY);
		const dim3 blocks(sizes[0] / threadsX, sizes[1] / threadsY);

		cv::Mat h_depthImage(cameraParams.depthImageHeight, cameraParams.depthImageWidth, CV_32FC1, (float *)depthMap);
		cv::cuda::GpuMat d_depthImage;
		d_depthImage.upload(h_depthImage);
		updateReconstructionKernel<<<blocks, threads>>>(
			model.gridSize,
			model.getGPUGrid(),
			cameraParams,
			d_depthImage,
			poseInverse,
			MINF);

		hipDeviceSynchronize();

		cv::Mat tempResult;

		// deviceModel.download(tempResult);
		// model.setGrid(tempResult);
		hipError_t err = hipGetLastError();
		if (err != hipSuccess)
		{
			printf("CUDA Error: %s\n", hipGetErrorString(err));
			// Possibly: exit(-1) if program cannot continue....
		}
	}

	void rayCast(Volume &model,
				 const CameraParameters &cameraParams,
				 const MatrixXf &cameraPose)
	{
		std::cout << "raycasting" << std::endl;
		cv::Mat surfacePoints(cameraParams.depthImageHeight, cameraParams.depthImageWidth, CV_32FC3),
			surfaceNormals(cameraParams.depthImageHeight, cameraParams.depthImageWidth, CV_32FC3);
		surfacePoints.setTo(0);
		surfaceNormals.setTo(0);

		cv::cuda::GpuMat deviceModel, deviceSurfacePoints, deviceSurfaceNormals; //(sizes,CV_32FC2);

		// TODO: Find better optimization for GPU Arch
		const int threadsX = 1, threadsY = 1;
		const dim3 threads(threadsX, threadsY);
		const dim3 blocks(cameraParams.depthImageWidth / threadsX, cameraParams.depthImageHeight / threadsY);

		// deviceModel.upload(model.getGrid());
		deviceSurfacePoints.upload(surfacePoints);
		deviceSurfaceNormals.upload(surfaceNormals);

		rayCastKernel<<<blocks, threads>>>(
			cameraPose,
			cameraParams,
			model.gridSize,
			model.getGPUGrid(),
			deviceSurfacePoints,
			deviceSurfaceNormals);

		hipDeviceSynchronize();

		hipError_t err = hipGetLastError();
		if (err != hipSuccess)
		{
			printf("CUDA Error: %s\n", hipGetErrorString(err));
			// Possibly: exit(-1) if program cannot continue....
		}
		deviceSurfacePoints.download(surfacePoints);
		deviceSurfaceNormals.download(surfaceNormals);

		std::vector<Vector3f> points, normals;
		for (int i = 0; i < cameraParams.depthImageHeight; ++i)
		{
			for (int j = 0; j < cameraParams.depthImageWidth; ++j)
			{

				if (!(surfacePoints.at<cv::Vec3f>(i, j)[0] == 0 &&
					  surfacePoints.at<cv::Vec3f>(i, j)[1] == 0 &&
					  surfacePoints.at<cv::Vec3f>(i, j)[2] == 0))
				{
					points.push_back(Vector3f(surfacePoints.at<cv::Vec3f>(i, j)[0],
											  surfacePoints.at<cv::Vec3f>(i, j)[1],
											  surfacePoints.at<cv::Vec3f>(i, j)[2]));

					normals.push_back(Vector3f(surfaceNormals.at<cv::Vec3f>(i, j)[0],
											   surfaceNormals.at<cv::Vec3f>(i, j)[1],
											   surfaceNormals.at<cv::Vec3f>(i, j)[2]));
				}
			}
		}
		static int shitCounter = 0;

		cv::imwrite("DepthImage" + std::to_string(shitCounter++) + ".png", (surfaceNormals + 1.0f) / 2.0 * 255.0f);
		model.setSurfaceNormals(surfaceNormals);
		model.setSurfacePoints(surfacePoints);
		PointCloud pcd(points, normals);
		// pcd.writeMesh("plsowrk" + std::to_string(shitCounter++) + ".off");
		model.setPointCloud(pcd);
	}

	
	void poseEstimation(Matrix4f &modelToFramePose, const CameraParameters &cameraParams, cv::cuda::GpuMat surfacePoints, cv::cuda::GpuMat surfaceNormals,
						PointCloud &inputPCD) // c// cv::cuda::GpuMat newVertexMap, cv::cuda::GpuMat newNormalMap)
	{

		const int threadsX = 1, threadsY = 1;
		const dim3 threads(threadsX, threadsY);
		const dim3 blocks(cameraParams.depthImageWidth /
							  threadsX,
						  cameraParams.depthImageHeight / threadsY);

		//Compute pcd vertices and normals as opencv MAT and send them to gpu

		cv::cuda::GpuMat newVertexMap;
		cv::cuda::GpuMat newNormalMap;
		cv::Mat hostVertexMap(cameraParams.depthImageHeight, cameraParams.depthImageWidth, CV_32FC3);
		cv::Mat hostNormalMap(cameraParams.depthImageHeight, cameraParams.depthImageWidth, CV_32FC3);
		cv::Mat sourceMap(cameraParams.depthImageHeight, cameraParams.depthImageWidth, CV_32FC3);;
		surfacePoints.download(sourceMap);
		int numPoints = inputPCD.getPoints().size();
		auto pts = inputPCD.getPoints();
		auto nrmls = inputPCD.getNormals();
		for (int i = 0; i < cameraParams.depthImageHeight; i++)
		{
			for (int j = 0; j < cameraParams.depthImageWidth; j++)
			{
				if (pts[i * cameraParams.depthImageWidth + i].x() != MINF)
				{
					auto pnt = pts[i * cameraParams.depthImageWidth + i];
					auto normal = nrmls[i * cameraParams.depthImageWidth + i];

					hostVertexMap.at<cv::Vec3f>(i, j)[0] = pnt.x();
					hostVertexMap.at<cv::Vec3f>(i, j)[1] = pnt.y();
					hostVertexMap.at<cv::Vec3f>(i, j)[2] = pnt.z();

					hostNormalMap.at<cv::Vec3f>(i, j)[0] = normal.x();
					hostNormalMap.at<cv::Vec3f>(i, j)[1] = normal.y();
					hostNormalMap.at<cv::Vec3f>(i, j)[2] = normal.z();
				}
			}
		}		
		newVertexMap.upload(hostVertexMap);
		newNormalMap.upload(hostNormalMap);
		cv::cuda::GpuMat matches;
		cv::Mat hostMatches(cameraParams.depthImageHeight, cameraParams.depthImageWidth, CV_32SC2);
		MatrixXf estimatedCameraPose = modelToFramePose;		   //initial
		MatrixXf modelToFrameInverse = modelToFramePose.inverse(); //previous frame to model
		matches.setTo(0);
		hostMatches.setTo(0);
		for (int i = 0; i < ICP_ITERATIONS; i++)
		{
			hostMatches.setTo(0);
			matches.upload(hostMatches);
			hipDeviceSynchronize();

			findCorrespondencesKernel<<<blocks, threads>>>(modelToFrameInverse,
														   estimatedCameraPose,
														   cameraParams,
														   surfacePoints,
														   surfaceNormals,
														   newVertexMap,
														   newNormalMap,
														   matches);
			hipDeviceSynchronize();

			hipError_t err = hipGetLastError();
			matches.download(hostMatches);
			cv::Mat splittedMatches[3];
			cv::split(hostMatches, splittedMatches);

			int nzCount = cv::countNonZero(splittedMatches[0]);
			std::cout << nzCount << std::endl;
			std::vector<Vector3f> sourcePts;
			std::vector<Vector3f> targetPts;
			std::vector<Vector3f> targetNormals;
			Matrix3f rotation = estimatedCameraPose.block<3, 3>(0, 0);
			Vector3f translation = estimatedCameraPose.block<3, 1>(0, 3);
			for (int i = 0; i < cameraParams.depthImageHeight; i++)
			{
				for (int j = 0; j < cameraParams.depthImageWidth; j++)
				{
					if (hostMatches.at<cv::Vec2i>(i, j)[0] != 0 && hostMatches.at<cv::Vec2i>(i, j)[1] != 0)
					{
						int x = hostMatches.at<cv::Vec2i>(i, j)[0];
						int y = hostMatches.at<cv::Vec2i>(i, j)[1];
						Vector3f pnt;
						pnt.x()= hostVertexMap.at<cv::Vec3f>(i,j)[0];
						pnt.y()= hostVertexMap.at<cv::Vec3f>(i,j)[1];
						pnt.z()= hostVertexMap.at<cv::Vec3f>(i,j)[2];
						Vector3f normal;
						normal.x()= hostNormalMap.at<cv::Vec3f>(i,j)[0];
						normal.y()= hostNormalMap.at<cv::Vec3f>(i,j)[1];
						normal.z()= hostNormalMap.at<cv::Vec3f>(i,j)[2];

						targetPts.push_back(pnt);
						targetNormals.push_back(normal);
						Vector3f srcPoint(sourceMap.at<cv::Vec3f>(i, j)[0], sourceMap.at<cv::Vec3f>(i, j)[1], sourceMap.at<cv::Vec3f>(i, j)[2]);
						sourcePts.push_back(rotation * srcPoint + translation);

						// printf("source %d %d --> target %d %d \n",i,j,x,y);
					}
				}
			}

			if (err != hipSuccess)
			{
				printf("CUDA Error: %s\n", hipGetErrorString(err));
				// Possibly: exit(-1) if program cannot continue....
			}
							//estimatedPose = estimatePosePointToPlane(sourcePoints, targetPoints, target.getNormals()) * estimatedPose;

			estimatedCameraPose = estimatePosePointToPlane(sourcePts, targetPts, targetNormals)*estimatedCameraPose;
		}
		std::cout << modelToFramePose << std::endl;
		std::cout << "***************"<< std::endl;
		std::cout << estimatedCameraPose<< std::endl;
		modelToFramePose = estimatedCameraPose;

	}
	Matrix4f estimatePosePointToPlane(const std::vector<Vector3f> &sourcePoints, const std::vector<Vector3f> &targetPoints, const std::vector<Vector3f> &targetNormals)
	{
		const unsigned nPoints = sourcePoints.size();

		// Build the system
		MatrixXf A = MatrixXf::Zero(4 * nPoints, 6);
		VectorXf b = VectorXf::Zero(4 * nPoints);

		for (unsigned i = 0; i < nPoints; i++)
		{
			const auto &s = sourcePoints[i];
			const auto &d = targetPoints[i];
			const auto &n = targetNormals[i];

			// TODO: [DONE] Add the point-to-plane constraints to the system
			//  1 point-to-plane row per point
			A(4 * i, 0) = n.z() * s.y() - n.y() * s.z();
			A(4 * i, 1) = n.x() * s.z() - n.z() * s.x();
			A(4 * i, 2) = n.y() * s.x() - n.x() * s.y();
			A(4 * i, 3) = n.x();
			A(4 * i, 4) = n.y();
			A(4 * i, 5) = n.z();
			b(4 * i) = n.x() * d.x() + n.y() * d.y() + n.z() * d.z() - n.x() * s.x() - n.y() * s.y() - n.z() * s.z();

			// TODO: [DONE] Add the point-to-point constraints to the system
			//  3 point-to-point rows per point (one per coordinate)
			A(4 * i + 1, 0) = 0.0f;
			A(4 * i + 1, 1) = s.z();
			A(4 * i + 1, 2) = -s.y();
			A(4 * i + 1, 3) = 1.0f;
			A(4 * i + 1, 4) = 0.0f;
			A(4 * i + 1, 5) = 0.0f;
			b(4 * i + 1) = d.x() - s.x();

			A(4 * i + 2, 0) = -s.z();
			A(4 * i + 2, 1) = 0.0f;
			A(4 * i + 2, 2) = s.x();
			A(4 * i + 2, 3) = 0.0f;
			A(4 * i + 2, 4) = 1.0f;
			A(4 * i + 2, 5) = 0.0f;
			b(4 * i + 2) = d.y() - s.y();

			A(4 * i + 3, 0) = s.y();
			A(4 * i + 3, 1) = -s.x();
			A(4 * i + 3, 2) = 0.0f;
			A(4 * i + 3, 3) = 0.0f;
			A(4 * i + 3, 4) = 0.0f;
			A(4 * i + 3, 5) = 1.0f;
			b(4 * i + 3) = d.z() - s.z();

			// TODO: [DONE] Optionally, apply a higher weight to point-to-plane correspondences
			float LAMBDA_plane = 1.0f;
			float LAMBDA_point = 0.1f;
			A(4 * i) *= LAMBDA_plane;
			b(4 * i) *= LAMBDA_plane;

			A(4 * i + 1) *= LAMBDA_point;
			b(4 * i + 1) *= LAMBDA_point;
			A(4 * i + 2) *= LAMBDA_point;
			b(4 * i + 2) *= LAMBDA_point;
			A(4 * i + 3) *= LAMBDA_point;
			b(4 * i + 3) *= LAMBDA_point;
		}

		// TODO: [DONE] Solve the system
		VectorXf x(6);

		JacobiSVD<MatrixXf> svd(A, ComputeThinU | ComputeThinV);
		const MatrixXf &E_i = svd.singularValues().asDiagonal().inverse();
		const MatrixXf &U_t = svd.matrixU().transpose();
		const MatrixXf &V = svd.matrixV();

		x = V * E_i * U_t * b;

		float alpha = x(0), beta = x(1), gamma = x(2);

		// Build the pose matrix
		Matrix3f rotation = AngleAxisf(alpha, Vector3f::UnitX()).toRotationMatrix() *
							AngleAxisf(beta, Vector3f::UnitY()).toRotationMatrix() *
							AngleAxisf(gamma, Vector3f::UnitZ()).toRotationMatrix();

		Vector3f translation = x.tail(3);

		// TODO: [DONE] Build the pose matrix using the rotation and translation matrices
		Matrix4f estimatedPose = Matrix4f::Identity();
		estimatedPose.block<3, 3>(0, 0) = rotation;
		estimatedPose.block<3, 1>(0, 3) = translation;

		return estimatedPose;
	}
} // namespace Wrapper