#include "hip/hip_runtime.h"
#include "dummy.cuh"
#include <iostream>
#include <opencv2/core/cuda.hpp>
#include "cuda_error_handle.h"
#include "Volume.h"

__global__ void test_kernel(cv::cuda::PtrStepSzf volume)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	// unsigned int index = row * (img.cols) + col;
	int dim = 512; // TODO: pass this as parameter;
	unsigned int lim = dim * dim * dim;
	if (x >= 0 && x < dim && y >= 0 && y < dim)
	{
		for (auto z = 0; z < dim; z++)
		{
			int ind = (x * dim + y) * dim + z;
			if (ind < lim)
			{
				volume(ind, 0) = x - y;
				volume(ind, 1) = z;
			}
			// }
			// volume.ptr(x,y,z)[1] = 0;
			// volume.at<cv::Vec2f>(x,y,z)[0] = 0;
			// volume.at<cv::Vec2f>(x,y,z)[1] = 0;
		}
	}
	// *i = 99;
}

namespace Wrapper
{
	void wrapper(cv::cuda::GpuMat &img, Volume &model)
	{
		std::vector<int> sizes{512, 512, 512};
		cv::cuda::GpuMat deviceModel; //(sizes,CV_32FC2);

		// TODO: Find better optimization for GPU Arch
		const dim3 threads(32, 32);
		const dim3 blocks(512 / 32, 512 / 32);

		// ! TODO: First element may overflow
		std::vector<int> flattenedSize{
			model.gridSize.x() * model.gridSize.y() * model.gridSize.z(),
			1
		};
		deviceModel.upload(model.getGrid().reshape(2, flattenedSize));

		hipError_t err = hipGetLastError();

		test_kernel<<<blocks, threads>>>(deviceModel);
		hipDeviceSynchronize();

		cv::Mat tempResult;
		deviceModel.download(tempResult);
		model.getGrid() = tempResult.reshape(2, sizes);
		// model.setGrid(m);

		err = hipGetLastError();

		if (err != hipSuccess)
		{
			printf("CUDA Error: %s\n", hipGetErrorString(err));

			// Possibly: exit(-1) if program cannot continue....
		}
		// d_img_ = d_img_original;
		hipDeviceSynchronize();
	}
} // namespace Wrapper